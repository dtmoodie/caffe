#include "hip/hip_runtime.h"
#include "caffe/layers/seginfogain_loss_layer.hpp"
#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#include "hip/hip_runtime.h"

#define CUDA_KERNEL_LOOP_N(i, n, Dim) \
  for (int i = blockIdx.Dim * blockDim.Dim + threadIdx.Dim; \
       i < (n); \
       i += blockDim.Dim * gridDim.Dim)

template<typename Dtype>
void __global__  seginfo_gain_loss_forward_kernel(const Dtype* label, const int max_label,
                                         const int ignore_label, const int numLabels,
                                         const int outer_num_, const int inner_num_,
                                         const Dtype* infogain_mat, Dtype* loss, Dtype* count,
                                                  const Dtype* prob_data, const int dim)
{
    CUDA_KERNEL_LOOP(i, outer_num_)
    {
        for(int j = 0; j < inner_num_; ++j)
        {
            const int label_value = static_cast<int>(label[i * inner_num_ + j]);
            if(label_value == ignore_label ||
                label_value < 0 || label_value >= max_label)
                continue;
            for (int k = 0; k < numLabels; k++)
            {
                atomicAdd(loss,- (infogain_mat[label_value * numLabels + k] *
                        log(max(prob_data[i * dim + k * inner_num_ + j],
                            Dtype(caffe::kLOG_THRESHOLD)))));
            }
            atomicAdd(count, (Dtype)1.0);
        }
    }
}

template<typename Dtype>
void __global__ seginfo_gain_loss_backwards_kernel(const Dtype* label, const int numLabels,
                                                   const int ignore_label_, Dtype* bottom_diff,
                                                   const Dtype* infogain_mat,
                                                   const Dtype* infogain_sum, Dtype* count,
                                                   const int channels,
                                                   const int outer_num_, const int inner_num_,
                                                   const int dim)
{

    CUDA_KERNEL_LOOP(i, outer_num_) {
        for (int j = 0; j < inner_num_; ++j) {
            const int label_value = static_cast<int>(label[i * inner_num_ + j]);
            if (label_value == ignore_label_) {
                for (int c = 0; c < channels; ++c) {
                    bottom_diff[i * dim + c * inner_num_ + j] = 0;
                }
            }
            else {

                for (int k = 0; k < numLabels; k++)
                {
                    bottom_diff[i * dim + k * inner_num_ + j] *= infogain_sum[label_value];
                    bottom_diff[i * dim + k * inner_num_ + j] -= infogain_mat[label_value * numLabels + k];
                }
                //++count;
                atomicAdd(count, (Dtype)1.0);
            }
        }
    }
}

namespace caffe{
  template<typename Dtype>
  void SegInfogainLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top){
    softmax_layer_->Forward(bottom, top);
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* label = bottom[1]->gpu_data();

    const Dtype* infogain_mat = NULL;
    if (bottom.size() < 3) {
        infogain_mat = infogain_.gpu_data();
    }
    else {
        infogain_mat = bottom[2]->gpu_data();
    }
    int dim = prob_.count() / outer_num_; //step between cases: number of voxels*labels
    int numLabels = prob_.count() / outer_num_ / inner_num_;

    Dtype* loss = 0;
    Dtype* count = 0;
    hipMalloc(&loss, sizeof(Dtype));
    hipMemset(loss, 0, sizeof(Dtype));
    hipMalloc(&count, sizeof(Dtype));
    hipMemset(count, 0, sizeof(Dtype));
    seginfo_gain_loss_forward_kernel<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
            CAFFE_CUDA_NUM_THREADS>>>(label, prob_.shape(softmax_axis_),
                                               has_ignore_label_? ignore_label_ : -1,
                                               numLabels, outer_num_, inner_num_,
                                               infogain_mat, loss, count, prob_data, dim);
    hipFree(loss);
    hipFree(count);

    if (normalize_) {
        //top[0]->mutable_cpu_data()[0] = loss / count;
        caffe_gpu_div(1, loss, count, top[0]->mutable_gpu_data());
    }
    else {
        //top[0]->mutable_cpu_data()[0] = loss / outer_num_;
        caffe_gpu_scale(1, 1.0f / (Dtype)outer_num_, loss, top[0]->mutable_gpu_data());
    }
    if (top.size() == 2) {
        top[1]->ShareData(prob_);
    }
  }

  template<typename Dtype>
  void SegInfogainLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
      if (propagate_down[1]) {
          LOG(FATAL) << this->type()
              << " Layer cannot backpropagate to label inputs.";
      }
      if (propagate_down.size() > 2 && propagate_down[2]) {
          LOG(FATAL) << this->type()
              << " Layer cannot backpropagate to infogain inputs.";
      }
      if (propagate_down[0]) {
          Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
          const Dtype* prob_data = prob_.gpu_data();

          //caffe_gpu_copy(prob_.count(), prob_data, bottom_diff);
          hipMemcpy(bottom_diff, prob_data, prob_.count() * sizeof(Dtype), hipMemcpyDeviceToDevice);


          const Dtype* label = bottom[1]->gpu_data();
          const Dtype* infogain_mat = NULL;
          Dtype* infogain_sum = infogain_sum_.mutable_gpu_data();

          int numLabels = prob_.count() / outer_num_ / inner_num_;
          int dim = prob_.count() / outer_num_;
          if (bottom.size() < 3) {
              infogain_mat = infogain_.gpu_data();
          }
          else {
              infogain_mat = bottom[2]->gpu_data();
              for (int labelIt = 0; labelIt < numLabels; labelIt++)
              {
                  //infogain_sum[labelIt] = caffe_gpu_asum(numLabels, infogain_mat + labelIt * numLabels);
                  caffe_gpu_asum(numLabels, infogain_mat + labelIt * numLabels, infogain_sum + labelIt);
              }
          }
          const Dtype* infogainSum = infogain_sum_.gpu_data();
          Dtype* count = 0;
          hipMalloc(&count, sizeof(Dtype));
          hipMemset(count, 0, sizeof(Dtype));
          seginfo_gain_loss_backwards_kernel<<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
                  CAFFE_CUDA_NUM_THREADS>>>(label, numLabels, has_ignore_label_ ? ignore_label_ : -1,
                                             bottom_diff, infogain_mat, infogainSum, count, bottom[0]->shape(softmax_axis_),
                                            outer_num_, inner_num_, dim);

          // Scale gradient
          const Dtype loss_weight = top[0]->cpu_diff()[0];
          if (normalize_) {
              //caffe_scal(prob_.count(), loss_weight / count, bottom_diff);
              Dtype h_count;
              hipMemcpy(&h_count, count, sizeof(Dtype), hipMemcpyDeviceToHost);
              caffe_gpu_scal(prob_.count(), loss_weight / h_count, bottom_diff);

          }
          else {
              //caffe_scal(prob_.count(), loss_weight / outer_num_, bottom_diff);
              caffe_gpu_scal(prob_.count(), loss_weight / outer_num_, bottom_diff);
          }
      }
  }
  INSTANTIATE_LAYER_GPU_FUNCS(SegInfogainLossLayer);
}



